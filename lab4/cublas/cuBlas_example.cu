#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define m 8192 // a - mxk matrix
#define n 8192 // b - kxn matrix
#define k 8192 // c - mxn matrix

void init(double* A, int t){
   for(int i=0;i<t*t;i++)
      A[i]=(rand()%10000+1)/1000.0;
}

int main (int argc, char** argv){
   hipError_t cudaStat ; // hipMalloc status
   hipblasStatus_t stat ; // CUBLAS functions status
   hipblasHandle_t handle ; // CUBLAS context
   time_t t;

   float milliseconds = 0;
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   // int i,j; // i-row index ,j- column index
   double * a; // mxk matrix a on the host
   double * b; // kxn matrix b on the host
   double * c; // mxn matrix c on the host

   /* Intializes random number generator */
   srand((unsigned) time(&t));

   a=(double *) malloc (m*k* sizeof (double)); // host memory for a
   b=(double *) malloc (k*n* sizeof (double)); // host memory for b
   c=(double *) malloc (m*n* sizeof (double)); // host memory for c

   // define an mxk matrix a column by column
   // int ind =11; // a:
   // for(j=0;j<k;j++){ // 11 ,17 ,23 ,29 ,35
   //    for(i=0;i<m;i++){ // 12 ,18 ,24 ,30 ,36
   //       a[IDX2C(i,j,m)]=(double)ind++; // 13 ,19 ,25 ,31 ,37
   //    } // 14 ,20 ,26 ,32 ,38
   // } // 15 ,21 ,27 ,33 ,39


   // 16 ,22 ,28 ,34 ,40
   // print a row by row
   // printf ("a:\n");
   // for(i=0;i<m;i++){
   //    for(j=0;j<k;j++){
   //       printf (" %f",a[ IDX2C (i,j,m )]);
   //    }
   //    printf ("\n");
   // }

   // define a kxn matrix b column by column
   // ind =11; // b:
   // for(j=0;j<n;j ++){ // 11 ,16 ,21 ,26
   //    for(i=0;i<k;i ++){ // 12 ,17 ,22 ,27
   //       b[IDX2C(i,j,k)]=(double)ind ++; // 13 ,18 ,23 ,28
   //    } // 14 ,19 ,24 ,29
   // } // 15 ,20 ,25 ,30

   // print b row by row
   // printf ("b:\n");
   // for(i=0;i<k;i++){
   //    for(j=0;j<n;j++){
   //       printf (" %f",b[IDX2C(i,j,k)]);
   //    }
   //    printf ("\n");
   // }

   // define an mxn matrix c column by column
   // ind =11; // c:
   // for(j=0;j<n;j++){ // 11 ,17 ,23 ,29
   //    for(i=0;i<m;i++){ // 12 ,18 ,24 ,30
   //       c[IDX2C(i,j,m)]=(double)ind ++; // 13 ,19 ,25 ,31
   //    } // 14 ,20 ,26 ,32
   // } // 15 ,21 ,27 ,33

   // 16 ,22 ,28 ,34
   // print c row by row
   // printf ("c:\n");
   // for(i=0;i<m;i++){
   //    for(j=0;j<n;j++){
   //       printf (" %f",c[ IDX2C (i,j,m )]);
   //    }
   //    printf ("\n");
   // }

   init(a,m);
   init(b,n);

   // on the device
   double * d_a; // d_a - a on the device
   double * d_b; // d_b - b on the device
   double * d_c; // d_c - c on the device

   cudaStat = hipMalloc (( void **)& d_a ,m*k* sizeof (*a)); // device memory alloc for a
   cudaStat = hipMalloc (( void **)& d_b ,k*n* sizeof (*b)); // device memory alloc for b
   cudaStat = hipMalloc (( void **)& d_c ,m*n* sizeof (*c)); // device memory alloc for c

   stat = hipblasCreate (& handle ); // initialize CUBLAS context
   // copy matrices from the host to the device
   stat = hipblasSetMatrix (m,k, sizeof (*a) ,a,m,d_a ,m); //a -> d_a
   stat = hipblasSetMatrix (k,n, sizeof (*b) ,b,k,d_b ,k); //b -> d_b
   stat = hipblasSetMatrix (m,n, sizeof (*c) ,c,m,d_c ,m); //c -> d_c

   double al =1.0; // al =1
   double bet =1.0; // bet =1

   // matrix - matrix multiplication : d_c = al*d_a *d_b + bet *d_c
   // d_a -mxk matrix , d_b -kxn matrix , d_c -mxn matrix ;
   // al ,bet -scalars
   hipEventRecord(start);
   stat=hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,d_a,m,d_b,k,&bet,d_c,m);
   hipEventRecord(stop);

   stat = hipblasGetMatrix (m,n,sizeof(*c),d_c,m,c,m); // cp d_c - >c
   hipEventSynchronize(stop);
   // printf ("c after Dgemm :\n");
   // for(i=0;i<m;i ++){
   //    for(j=0;j<n;j ++){
   //       printf (" %f",c[IDX2C(i,j,m)]); // print c after Sgemm
   //    }
   //    printf ("\n");
   // }

   hipFree(d_a); // free device memory
   hipFree(d_b); // free device memory
   hipFree(d_c); // free device memory

   hipEventElapsedTime(&milliseconds, start, stop);
   printf("Execution time=%f ms, Gflops=%f\n", milliseconds, 2.0*pow(m,3)/milliseconds/1e6);
   fflush(stdout);

   hipblasDestroy ( handle ); // destroy CUBLAS context

   free (a); // free host memory
   free (b); // free host memory
   free (c); // free host memory

   return EXIT_SUCCESS ;
}
