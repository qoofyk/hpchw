#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
	int width;
	int height;
	double* elements;
} Matrix;

typedef struct gv_t {
	int N; //N*N matrix
	int loop;
	int block_size; // Thread block size
}* GV;

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
double MatMul(GV gv, const Matrix A, const Matrix B, Matrix C) {
	float milliseconds = 0;
  	hipEvent_t start, stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);

	// Load A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(double); hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(double);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

	// Allocate C in device memory
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = C.width * C.height * sizeof(double);
	hipMalloc(&d_C.elements, size);

	// Invoke kernel
	dim3 dimBlock(gv->block_size, gv->block_size);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

	hipEventRecord(start);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipEventRecord(stop);

	// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);

	// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);

	hipEventElapsedTime(&milliseconds, start, stop);

	// printf("Execution time=%f ms, Gflops=%f\n", milliseconds, 2.0*pow(gv->N,3)/milliseconds/1e6);
	// fflush(stdout);

	return milliseconds;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	double Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e]
				* B.elements[e * B.width + col];
	C.elements[row * C.width + col] = Cvalue;
}

void init(double* A, int n){
	for(int i=0;i<n*n;i++)
		A[i]=(rand()%10000+1)/1000.0;
}

/*init every element to 0*/
void init_0(double* C, int n){
	for(int i=0;i<n*n;i++)
		C[i]=0;
}

/*init every element to 1*/
void init_1(double* C, int n){
	for(int i=0;i<n*n;i++)
		C[i]=1;
}

/*verify the calculation whether equal to verify matrix*/
void verification(GV gv, double *C, int n){
	int i;
	for(i=0;i<n*n;i++){
		if(C[i] != gv->N){
			printf("i=%d, C[i]=%f Get Wrong result!\n",i,C[i]);
			fflush(stdout);
		}
	}
}

void check_malloc(void * pointer){
  if (pointer == NULL) {
    perror("Malloc error!\n");
    fprintf (stderr, "at %s, line %d.\n", __FILE__, __LINE__);
    exit(1);
  }
}

int main(int argc, char** argv) {
	Matrix A, B, C;
	double elapsed_time;
	time_t t;
	int i;
	GV gv;
	// double *B;
	// double a_p[] = {1,5,9,13,2,6,10,14,3,7,11,15,4,8,12,16};
	// double b_p[] = {16,12,8,4,15,11,7,3,14,10,6,2,13,9,5,1};


	if(argc != 4) {
		fprintf(stderr, "Usage: %s real_matrix_size, thread block_size\n", argv[0]);
		exit(1);
	}

	gv    = (GV) malloc(sizeof(*gv));

	gv->N = atoi(argv[1]);
	gv->block_size = atoi(argv[2]);
	gv->loop = atoi(argv[3]);
	printf("N=%d,block_size=%d,loop=%d\n",
		gv->N,gv->block_size,gv->loop);

	/* Intializes random number generator */
   	srand((unsigned) time(&t));

   	/*Initialise matrix A, B and verify matrix*/
 	//printf("Initialise A B matrix\n");
	// printf("-----------------------------\n");
	// fflush(stdout);
	A.width = gv->N;
	A.height = gv->N;
	A.elements = (double *)malloc(sizeof(double)*(gv->N*gv->N));
	check_malloc(A.elements);

	B.width = gv->N;
	B.height = gv->N;
	B.elements = (double *)malloc(sizeof(double)*(gv->N*gv->N));
	check_malloc(B.elements);

	C.width = gv->N;
	C.height = gv->N;
	C.elements = (double *)malloc(sizeof(double)*(gv->N*gv->N));
	check_malloc(C.elements);

	// verify 1
	init_1(A.elements,gv->N);
	init_1(B.elements,gv->N);
	init_0(C.elements,gv->N);
	MatMul(gv,A,B,C);
	verification(gv,C.elements,gv->N);
	printf("Pass VERIFY 1\n");
	fflush(stdout);

	// start calculation
	init(A.elements,gv->N);
	init(B.elements,gv->N);
	elapsed_time=0;
	for (i = 0; i < gv->loop; ++i){
		init_0(C.elements,gv->N);
		elapsed_time += MatMul(gv,A,B,C);
	}
	elapsed_time = elapsed_time/gv->loop;
	printf("CALCULATION: Naive GPU average elapsed time: %f seconds, Gflops=%f\n",
		elapsed_time/1e3, 2.0*pow(gv->N,3)/elapsed_time/1e6);

	free(A.elements);
	free(B.elements);
	free(C.elements);

	free(gv);

	return 0;
}
