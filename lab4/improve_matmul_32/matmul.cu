#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
// Thread block size
#define BLOCK_SIZE 32
// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride;
    double* elements;
} Matrix;

typedef struct gv_t {
    int N; //N*N matrix
    int loop;
}* GV;

// Get a matrix element
__device__ double GetElement(const Matrix A, int row, int col){
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, double value){
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col){
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}


// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
double MatMul(GV gv, const Matrix A, const Matrix B, Matrix C){
    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(double);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(double);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
    hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(double);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

    hipEventRecord(start);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    hipEventRecord(stop);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time=%f ms, Gflops=%f\n", milliseconds, 2.0*pow(gv->N,3)/milliseconds/1e6);
    fflush(stdout);

    return milliseconds;
}

// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C){
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
    // Each thread computes one element of Csub by accumulating results into Cvalue
    double Cvalue = 0;
    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;
    // Loop over all the sub-matrices of A and B that are required to compute Csub
    // Multiply each pair of sub-matrices together and accumulate the results
    #pragma unroll
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);
        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);
        // Shared memory used to store Asub and Bsub respectively
        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];
        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);
        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        #pragma unroll
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];
        // Synchronize to make sure that the preceding computation is done
        // before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    // Write Csub to device memory. Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}


void init(double* A, int n){
    for(int i=0;i<n*n;i++)
        A[i]=(rand()%10000+1)/1000.0;
}

/*init every element to 0*/
void init_0(double* C, int n){
    for(int i=0;i<n*n;i++)
        C[i]=0;
}

/*init every element to 1*/
void init_1(double* C, int n){
    for(int i=0;i<n*n;i++)
        C[i]=1;
}

/*verify the calculation whether equal to verify matrix*/
void verification(GV gv, double *C, int n){
    int i;
    for(i=0;i<n*n;i++){
        if(C[i] != gv->N){
            printf("i=%d, C[i]=%f Get Wrong result!\n",i,C[i]);
            fflush(stdout);
        }
    }
}

void check_malloc(void * pointer){
  if (pointer == NULL) {
    perror("Malloc error!\n");
    fprintf (stderr, "at %s, line %d.\n", __FILE__, __LINE__);
    exit(1);
  }
}

void show_element(GV gv, double* C, int n){
    int i;
    for(i=0;i<n;i++){
        printf("%f ", C[i]);
        fflush(stdout);
    }
    printf("\n");
}

int main(int argc, char** argv) {
    Matrix A, B, C;
    double elapsed_time;
    time_t t;
    int i;
    GV gv;
    // double *B;
    // double a_p[] = {1,5,9,13,2,6,10,14,3,7,11,15,4,8,12,16};
    // double b_p[] = {16,12,8,4,15,11,7,3,14,10,6,2,13,9,5,1};


    if(argc != 3) {
        fprintf(stderr, "Usage: %s real_matrix_size, loop\n", argv[0]);
        exit(1);
    }

    gv    = (GV) malloc(sizeof(*gv));

    gv->N = atoi(argv[1]);
    gv->loop = atoi(argv[2]);
    printf("N=%d,loop=%d\n", gv->N,gv->loop);

    /* Intializes random number generator */
    srand((unsigned) time(&t));

    /*Initialise matrix A, B and verify matrix*/
    //printf("Initialise A B matrix\n");
    // printf("-----------------------------\n");
    // fflush(stdout);
    A.width = A.height = A.stride = gv->N;
    A.elements = (double *)malloc(sizeof(double)*(gv->N*gv->N));
    check_malloc(A.elements);

    B.width = B.height = B.stride=gv->N;
    B.elements = (double *)malloc(sizeof(double)*(gv->N*gv->N));
    check_malloc(B.elements);

    C.width = C.height = C.stride=gv->N;
    C.elements = (double *)malloc(sizeof(double)*(gv->N*gv->N));
    check_malloc(C.elements);

    // verify 1
    init_1(A.elements,gv->N);
    // show_element(gv,A.elements,gv->N*gv->N);
    init_1(B.elements,gv->N);
    // show_element(gv,B.elements,gv->N*gv->N);
    init_0(C.elements,gv->N);
    elapsed_time = MatMul(gv,A,B,C);
    // show_element(gv,C.elements,gv->N*gv->N);
    verification(gv,C.elements,gv->N);
    printf("Pass VERIFY 1\n");
    fflush(stdout);

    // start calculation
    init(A.elements,gv->N);
    init(B.elements,gv->N);
    elapsed_time=0;
    for (i = 0; i < gv->loop; ++i){
        init_0(C.elements,gv->N);
        elapsed_time += MatMul(gv,A,B,C);
    }
    elapsed_time = elapsed_time/gv->loop;
    printf("CALCULATION: Naive GPU average elapsed time: %f milliseconds, Gflops=%f\n",
        elapsed_time, 2.0*pow(gv->N,3)/elapsed_time/1e6);

    free(A.elements);
    free(B.elements);
    free(C.elements);

    free(gv);

    return 0;
}
